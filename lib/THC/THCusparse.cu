#include "THCusparse.h"
#include "THCGeneral.h"

struct cusparseMatDescr;
typedef struct cusparseMatDescr *hipsparseMatDescr_t;

static hipsparseOperation_t convertTransToCusparseOperation(char trans) {
  if (trans == 't') return HIPSPARSE_OPERATION_TRANSPOSE;
  else if (trans == 'n') return HIPSPARSE_OPERATION_NON_TRANSPOSE;
  else if (trans == 'c') return HIPSPARSE_OPERATION_CONJUGATE_TRANSPOSE;
  else {
    THError("trans must be one of: t, n, c");
    return HIPSPARSE_OPERATION_TRANSPOSE;
  }
}


/* Level 3 */
void THCusparse_Scsrmm2(THCState *state, char transa, char transb, int m, int n, int k, int nnz, float alpha, float *csrValA, int * csrRowPtrA, int * csrColIndA, float * B, int ldb, float beta, float *C, int ldc)
{
  hipsparseOperation_t opa = convertTransToCusparseOperation(transa);
  hipsparseOperation_t opb = convertTransToCusparseOperation(transb);

  if( (m <= INT_MAX) && (n <= INT_MAX) && (k <= INT_MAX)  && (nnz <= INT_MAX) && (ldb <= INT_MAX) && (ldc <= INT_MAX) )
  {
    int i_m = (int)m;
    int i_n = (int)n;
    int i_k = (int)k;
    int i_nnz = (int)nnz;
    int i_ldb = (int)ldb;
    int i_ldc = (int)ldc;

    hipsparseHandle_t handle = THCState_getCurrentCusparseHandle(state);
    hipsparseSetStream(handle, THCState_getCurrentStream(state));

    hipsparseMatDescr_t matDescr;

    THCusparseCheck(hipsparseCreateMatDescr(&matDescr));

    hipsparseSetMatType(matDescr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(matDescr, HIPSPARSE_INDEX_BASE_ZERO);

    THCusparseCheck(hipsparseScsrmm2(handle, opa, opb, i_m, i_n, i_k, i_nnz, &alpha, matDescr, csrValA, csrRowPtrA, csrColIndA, B, i_ldb, &beta, C, i_ldc));
    THCusparseCheck(hipsparseDestroyMatDescr(matDescr));
    return;
  }
  THError("Cusparse_csrmm2 only supports m, n, k, lda, ldb, ldc"
          "with the bound [val] <= %d", INT_MAX);
}

void THCusparse_Dcsrmm2(THCState *state, char transa, char transb, int m, int n, int k, int nnz, double alpha, double *csrValA, int * csrRowPtrA, int * csrColIndA, double * B, int ldb, double beta, double *C, int ldc)
{
  hipsparseOperation_t opa = convertTransToCusparseOperation(transa);
  hipsparseOperation_t opb = convertTransToCusparseOperation(transb);

  if( (m <= INT_MAX) && (n <= INT_MAX) && (k <= INT_MAX)  && (nnz <= INT_MAX) && (ldb <= INT_MAX) && (ldc <= INT_MAX) )
  {
    int i_m = (int)m;
    int i_n = (int)n;
    int i_k = (int)k;
    int i_nnz = (int)nnz;
    int i_ldb = (int)ldb;
    int i_ldc = (int)ldc;

    hipsparseMatDescr_t matDescr;

    THCusparseCheck(hipsparseCreateMatDescr(&matDescr));

    hipsparseSetMatType(matDescr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(matDescr, HIPSPARSE_INDEX_BASE_ZERO);

    hipsparseHandle_t handle = THCState_getCurrentCusparseHandle(state);
    hipsparseSetStream(handle, THCState_getCurrentStream(state));

    THCusparseCheck(hipsparseDcsrmm2(handle, opa, opb, i_m, i_n, i_k, i_nnz, &alpha, matDescr, csrValA, csrRowPtrA, csrColIndA, B, i_ldb, &beta, C, i_ldc));
    THCusparseCheck(hipsparseDestroyMatDescr(matDescr));
    return;
  }
  THError("Cusparse_csrmm2 only supports m, n, k, lda, ldb, ldc"
          "with the bound [val] <= %d", INT_MAX);
}

void THCusparse_Scsrmv(THCState *state, char transa, int m, int n, int nnz, float alpha, float *csrValA, int * csrRowPtrA, int * csrColIndA, float * B, float beta, float *C)
{
  hipsparseOperation_t opa = convertTransToCusparseOperation(transa);

  if( (m <= INT_MAX) && (n <= INT_MAX)  && (nnz <= INT_MAX) )
  {
    int i_m = (int)m;
    int i_n = (int)n;
    int i_nnz = (int)nnz;

    hipsparseHandle_t handle = THCState_getCurrentCusparseHandle(state);
    hipsparseSetStream(handle, THCState_getCurrentStream(state));

    hipsparseMatDescr_t matDescr;

    THCusparseCheck(hipsparseCreateMatDescr(&matDescr));

    hipsparseSetMatType(matDescr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(matDescr, HIPSPARSE_INDEX_BASE_ZERO);

    THCusparseCheck(hipsparseScsrmv(THCState_getCurrentCusparseHandle(state), opa, i_m, i_n, i_nnz, &alpha, matDescr, csrValA, csrRowPtrA, csrColIndA, B, &beta, C));
    THCusparseCheck(hipsparseDestroyMatDescr(matDescr));
    return;
  }
  THError("Cusparse_csrmv only supports m, n"
          "with the bound [val] <= %d", INT_MAX);
}

void THCusparse_Dcsrmv(THCState *state, char transa, int m, int n, int nnz, double alpha, double *csrValA, int * csrRowPtrA, int * csrColIndA, double * B, double beta, double *C)
{
  hipsparseOperation_t opa = convertTransToCusparseOperation(transa);

  if( (m <= INT_MAX) && (n <= INT_MAX)  && (nnz <= INT_MAX) )
  {
    int i_m = (int)m;
    int i_n = (int)n;
    int i_nnz = (int)nnz;

    hipsparseHandle_t handle = THCState_getCurrentCusparseHandle(state);
    hipsparseSetStream(handle, THCState_getCurrentStream(state));
 
    hipsparseMatDescr_t matDescr;
   
    THCusparseCheck(hipsparseCreateMatDescr(&matDescr));

    hipsparseSetMatType(matDescr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(matDescr, HIPSPARSE_INDEX_BASE_ZERO);

    THCusparseCheck(hipsparseDcsrmv(THCState_getCurrentCusparseHandle(state), opa, i_m, i_n, i_nnz, &alpha, matDescr, csrValA, csrRowPtrA, csrColIndA, B, &beta, C));
    THCusparseCheck(hipsparseDestroyMatDescr(matDescr));
    return;
  }
  THError("Cusparse_csrmv only supports m, n"
          "with the bound [val] <= %d", INT_MAX);
}
